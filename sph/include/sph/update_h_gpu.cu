#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2022 CSCS, ETH Zurich
 *               2022 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Smoothing length update on the GPU
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include "cstone/primitives/math.hpp"
#include "sph/sph_gpu.hpp"
#include "sph/kernels.hpp"

namespace sph
{

template<class Th>
__global__ void updateSmoothingLengthGpuKernel(GroupView grp, unsigned ng0, const unsigned* nc, Th* h)
{
    cstone::LocalIndex tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= grp.numGroups) { return; }

    auto bodyBegin = grp.groupStart[tid];
    auto bodyEnd   = grp.groupEnd[tid];

    for (auto i = bodyBegin; i < bodyEnd; ++i)
    {
        h[i] = updateH(ng0, nc[i], h[i]);
    }
}

template<class Th>
void updateSmoothingLengthGpu(const GroupView& grp, unsigned ng0, const unsigned* nc, Th* h)
{
    unsigned numThreads = 256;
    unsigned numBlocks  = cstone::iceil(grp.numGroups, 256);

    updateSmoothingLengthGpuKernel<<<numBlocks, numThreads>>>(grp, ng0, nc, h);
}

template void updateSmoothingLengthGpu(const GroupView& grp, unsigned ng0, const unsigned* nc, float* h);
template void updateSmoothingLengthGpu(const GroupView& grp, unsigned ng0, const unsigned* nc, double* h);

} // namespace sph